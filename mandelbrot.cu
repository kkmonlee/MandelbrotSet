#include <png.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

// Macro
#define cudacheck(call) \ {\
  hipError_t res = (call);\
  if (res != hipSuccess) {\
    const char* err_str = hipGetErrorString(res);\
    fprintf(stderr, "%s (%d): %s in %s", __FILE__, __LINE__, err_str, #call);\
    exit(-1);
  }
}

// time spent
double gpu_time = 0;

// computes number of threads
int divup(int x, int y) {
  return x / y + (x % y ? 1 : 0);
}
